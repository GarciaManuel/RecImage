/*
 * A01701414_lab2.cu
 *
 *  Created on: 11/14/2019
 *      Author: Manuel Garcia
 *		ID: A01701414
*/


#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <dirent.h>
#include <signal.h>
#include <string.h>
#include <sys/types.h>
#include <sys/stat.h>

#define ThreadsPerBlock 24
#define MaxBlocks 128

/*
Calculation of Matrix multiplciation using C++ CUDA
*/

__device__ void similar(int col, int row, float *d_matrizE, int mE_x, int mE_y, float *d_matrizD, int mD_x, int mD_y, float *d_result, int r_x, int r_y, float percentage_num){
    // Get our current index at the resultant matrix
    int index = col + row * r_x;

    //Indexes helpers for the original matrix value
    int current = 0;
    int current_x = 0;
    int current_y = 0;

    //Current value of a rotation at 0 degrees and 180º degrees
    float rotation = 0.0;
    float rotation3 = 0.0;

    d_result[index] = 0;

    //Indexes for the 180 degrees
    int x = mD_x;
    int y = mD_y;

    for(int i = 0; i < mD_y; i ++){
        //Indexes actualization for the 180 degrees
        y = y -1;
        x= mD_x;
        for(int j = 0; j < mD_x; j ++){
            //Get current index for the original matrix
            current_x = (col + j);
            current_y = (row + i);

            //Indexes actualization for the 180 degrees
            x = x - 1;
            if(current_x < mE_x && current_y <mE_y){

                current = current_x + (current_y*(mE_x));
                //Offset from the user
                float offset1 = d_matrizD[j + i*mD_x] * percentage_num;
                //If between offset add 1
                if (d_matrizE[current] >= (d_matrizD[j + i*mD_x] + (offset1)) && d_matrizE[current] <= (d_matrizD[j + i*mD_x] - (offset1))) {
                    rotation += 1;

                }

                //Offset from the user
                float offset2 = d_matrizD[x + y*mD_x] * percentage_num;
                //If between offset add 1
                if (d_matrizE[current] >= (d_matrizD[x + y*mD_x]) + (offset2) &&  d_matrizE[current] <= (d_matrizD[x + y*mD_x]) - (offset2) ) {
                    rotation3 += 1;

                }


            }

        }
    }
    rotation = rotation/(mD_x*mD_y);
    rotation3 = rotation3/(mD_x*mD_y);
    //Check each rotation and get the value of max one
    if(rotation3 > rotation)
        d_result[index] = rotation3;
    else
        d_result[index] = rotation;


}

__global__ void valid_similarity(float *d_matrizE, int mE_x, int mE_y, float *d_matrizD, int mD_x, int mD_y, float *d_result, int r_x, int r_y, float percentage_num){
    //Get threads location
    int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

    //If the thread is located in the bounds of the resultant matrix the call similar to fill each cell of the resultant matrix
	if(r_y > row && r_x > col){
        similar(col, row, d_matrizE, mE_x, mE_y, d_matrizD, mD_x, mD_y, d_result, r_x, r_y, percentage_num);
	}
}

//Function to fill randomly matrix with numbers up to 20
void fill_matriz(float *matriz, int x, int y){
    for(int i = 0; i<x; i++){
        for(int j = 0; j<y; j++){
            matriz[i + j*x] = rand()%20;
        }
    }
}

//Function to read each amtrix form a file and return the pointer
float *read_matrix(char *file, int *mx, int *my)
{


    FILE* fin;
    float *matriz;
    int x, y;

    fin = fopen(file, "r");
    if (fin == NULL)
    {
        printf("File open error..");
        exit(0);
    }

    //Read sizes of x and y
    fscanf(fin, "%i,", mx);
    fscanf(fin, "%i\n", my);

    x = *mx;
    y = *my;

    matriz = (float *)malloc(sizeof(float) * x * y);

    //Read each value
    for (int i = 0; i < x; i++)
    {
        for (int j = 0; j < y; j++)
        {
            fscanf(fin, "%f,", &matriz[i + j * x]);
        }
        fscanf(fin, "\n");
    }

    //close descriptor
    fclose(fin);

    return matriz;
}

//Function to print matrix
void print_matriz(float *matriz, int x, int y){
    printf("\n");
    for(int i = 0; i<x; i++){
        for(int j = 0; j<y; j++){
            printf("%.3f \t",matriz[i + j*x]);
        }
        printf("\n");
    }
}
//Function to print matrix 90 degrees
void print_rotated(float *matriz, int x, int y){
    printf("\nRotated \n");
    for(int j = y-1; j>=0; j--){
        for(int i = 0; i<x; i++){
            printf("%.3f \t",matriz[i + j*x]);
        }
        printf("\n");
    }
}

//Function to print matrix 180 degrees
void print_rotated2(float *matriz, int x, int y){
    printf("\nRotated2 \n");
    for(int i = x-1; i>=0; i--){
        for(int j = y-1; j>=0; j--){
            printf("%.3f \t",matriz[i + j*x]);
        }
        printf("\n");
    }
}

//Function to print matrix 270 degrees
void print_rotated3(float *matriz, int x, int y){
    printf("\nRotated \n");
    for(int j = 0; j<y; j++){
        for(int i = x-1; i>=0; i--){
            printf("%.3f \t",matriz[i + j*x]);
        }
        printf("\n");
    }
}

//Function to get max result from matrix
float similarity(float *matriz, int x, int y){
    float max_res = 0.0;
    for(int i = 0; i<x; i++){
        for(int j = 0; j<y; j++){
            if (max_res < matriz[i + j*x]){
                max_res = matriz[i + j*x];
            }
        }
    }
    return max_res;
}

int main(int argc, char *argv[]){
    srand(time(NULL));   // Initialization, should only be called once. for the random
    close(0);
    //Receive params
    char *image;
    char *find;
    char *percentage;
    char *show_matrix;
    if (argc != 5)
    {
        printf("usage: %s initial_image_name image_to_find_name percentage_of_error show_resultant_matrix\n", argv[0]);
        return -1;
    }

    image = argv[1];
    find = argv[2];
    percentage = argv[3];
    show_matrix = argv[4];

    //Convert string to float form argv
    float percentage_num = atof(percentage);

    //Convert string to int form argv
    int show = atoi(show_matrix);

    //Declare ponters of cuda and C
    float *matrizE, *matrizD, *result;
    float *d_matrizE, *d_matrizD, *d_result;

    //Sizes of E & D
    int mE_x, mE_y, mD_x, mD_y;

    //Read each matrix
    matrizE = read_matrix(image, &mE_x, &mE_y);
    printf("\nOriginal image size \nWidth: %i, Height:%i \n", mE_y, mE_x);

    matrizD = read_matrix(find, &mD_x, &mD_y);
    printf("Image to find size \nWidth : %i, Height:%i \n", mD_y, mD_x);


    // Size of resultant matrix
    int r_x = mE_x;
    int r_y = mE_y;

    //Define blocks
    //int NumBlocks = (ThreadsPerBlock + (MaxBlocks*MaxBlocks-1))/ThreadsPerBlock;
    int NumBlocks = (ThreadsPerBlock + (r_x - 1))/ThreadsPerBlock;

    //Malloc resultant matrixes
    result = (float *)malloc(sizeof(float)*r_x*r_y);

    printf("\n");

    //Malloc in cuda for passing matrixes
    hipMalloc((void**)&d_matrizE,sizeof(float)*mE_x*mE_y);
    hipMalloc((void**)&d_matrizD,sizeof(float)*mD_x*mD_y);
    hipMalloc((void**)&d_result,sizeof(float)*r_x*r_y);

    //Copy pointers of matrix e & d

    hipMemcpy(d_matrizE, matrizE,sizeof(float)*mE_x*mE_y, hipMemcpyHostToDevice);
    hipMemcpy(d_matrizD, matrizD,sizeof(float)*mD_x*mD_y, hipMemcpyHostToDevice);

    //Define sizes
    dim3 Blocks(NumBlocks,NumBlocks);
    dim3 Threads(ThreadsPerBlock,ThreadsPerBlock);


    printf("\nCalculating ... \n");

    //Take time for the cuda execution
    clock_t begin = clock();

    //Call Cuda global
    valid_similarity<<<Blocks, Threads>>>(d_matrizE, mE_x, mE_y, d_matrizD, mD_x, mD_y, d_result, r_x, r_y, percentage_num);

    clock_t end = clock();

    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    //Copy result
    hipMemcpy(result, d_result,sizeof(float)*r_x*r_y, hipMemcpyDeviceToHost);

    //Show result matrix if user wants
    printf("\n");
    if(show){
        printf("\nResultant matrix \n");
        print_matriz(result, r_x, r_y);

    }

    //Return value
    float percentage_similarity = similarity(result, r_x,r_y);
    printf("\nThe percentage of similarity with %s margin of error is: %f \n", argv[3], percentage_similarity);
    printf("Time spent: %f \n", time_spent);


    //Free all pointers
    free(matrizE);
    free(matrizD);
    free(result);

    //Free all pointers on device
    hipFree(d_matrizE);
    hipFree(d_matrizD);
    hipFree(d_result);

}

