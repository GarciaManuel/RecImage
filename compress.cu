/*
 * A01701414_lab2.cu
 *
 *  Created on: 11/14/2019
 *      Author: Manuel Garcia
 *		ID: A01701414
*/

#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h> 
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <dirent.h>
#include <signal.h>
#include <string.h>
#include <sys/types.h>
#include <sys/stat.h>

#define ThreadsPerBlock 25


/*
Calculation of Matrix multiplciation using C++ CUDA
*/

__device__ void compress(int col, int row, float *d_matrizE, int mE_x, int mE_y, int mD_x, int mD_y, float *d_result, int r_x, int r_y){
    // Get our current index at the resultant matrix
    int index = col + row * r_x;

    int index_x = 0;
    int index_y = 0;
    int current = 0;

    float promedio = 0.0;
    int count = 0;

    //Iterate in the image matrix
    for(int i = 0; i < mD_y; i++){
        for(int j = 0; j < mD_x; j++){
            index_x = col + j;
            index_y = row + i;

            //Eval if index is within bounds
            if(index_x < mE_x && index_y <mE_y){
                current = index_x + index_y * mE_x;
                count += 1;
                //add up all cells in compress dimension
                promedio += d_matrizE[current];
            }
        }
    }

    //Divide the result by the number of items summed and put into the resultant matrix.
    d_result[index] = promedio/count;
    
}

__global__ void valid_compression(float *d_matrizE, int mE_x, int mE_y, int mD_x, int mD_y, float *d_result, int r_x, int r_y){
    //Get threads location
    int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

    //If the thread is located in the bounds of the resultant matrix the call compress to fill each cell of the resultant matrix
	if(r_y > row && r_x > col){
        compress(col, row, d_matrizE, mE_x, mE_y, mD_x, mD_y, d_result, r_x, r_y);

	}
}

//Function to fill randomly matrix with numbers up to 20
void fill_matriz(float *matriz, int x, int y){
    for(int i = 0; i<x; i++){
        for(int j = 0; j<y; j++){
            matriz[i + j*x] = rand()%20;
        }
    }
}
//Function to read each amtrix form a file and return the pointer
float *read_matrix(char *file, int *mx, int *my)
{
    
    FILE* fin;
    float *matriz;
    int x, y;

    printf("Opening the file...\n");
    fin = fopen(file, "r");
    if (fin == NULL)
    {
        printf("File open error..");
        exit(0);
    }
    else
    {
        printf("File opened successfully..");
    }

    //Read sizes of x and y
    fscanf(fin, "%i,", mx);
    fscanf(fin, "%i\n", my);

    x = *mx;
    y = *my;

    matriz = (float *)malloc(sizeof(float) * x * y);

    //Read each value
    for (int i = 0; i < x; i++)
    {
        for (int j = 0; j < y; j++)
        {
            fscanf(fin, "%f,", &matriz[i + j * x]);
        }
        fscanf(fin, "\n");
    }

    //close descriptor
    fclose(fin);

    return matriz;
}
//Function to write a matrix into the file compressedImage.txt
void write_file(float *matrix, int x, int y){
    FILE* fin;

    printf("Opening the file...\n");
    fin = fopen("compressedImage.txt", "w+");
    if (fin == NULL)
    {
        printf("File open error..");
        exit(0);
    }
    else
    {
        printf("File opened successfully..");
    }
     //Print sizes of x and y
     fprintf(fin, "%i,", x);
     fprintf(fin, "%i\n", y);

    //Print each value
    for (int i = 0; i < x; i++)
    {
        for (int j = 0; j < y; j++)
        {
            fprintf(fin, "%i,", (int)matrix[i + j * x]);
        }
        fprintf(fin, "\n");
    }

    //close descriptor
    fclose(fin);

}

//Function to print matrix
void print_matriz(float *matriz, int x, int y){
    printf("\n");
    for(int i = 0; i<x; i++){
        for(int j = 0; j<y; j++){
            printf("%.1f \t",matriz[i + j*x]);
        }
        printf("\n");
    }
}

int main(int argc, char *argv[]){
    srand(time(NULL));   // Initialization, should only be called once. for the random
    close(0);
    //Receive params
    char *image;
    char *dim_x;
    char *dim_y;

    if (argc != 4)
    {
        printf("usage: %s initial_image_name dim_x dim_y \n", argv[0]);
        return -1;
    }
    image = argv[1];
    dim_x = argv[2];
    dim_y = argv[3];

    //Declare ponters of cuda and C
    float *matrizE, *result;
    float *d_matrizE, *d_result;

    //Sizes of E & D
    int mE_x, mE_y, mD_x, mD_y;

    //Get sizes of matrix compression
    mD_x = atoi(dim_x);
    mD_y = atoi(dim_y);

    //Read E matrix
    matrizE = read_matrix(image, &mE_x, &mE_y);
    printf("x : %i, y:%i \n", mE_x, mE_y);

    // Size of resultant matrix
    int r_x = 1 + ((mE_x - 1) / mD_x);
    int r_y = 1 + ((mE_y - 1) / mD_y);
    
    //Define blocks
    int NumBlocks = (ThreadsPerBlock + (r_x - 1))/ThreadsPerBlock;

    //Malloc matrixes
    result = (float *)malloc(sizeof(float)*r_x*r_y);

    printf("Compress image \n");

    printf("Originals \nMatrix E:\n");

    printf("Size x : %i, size y: %i \n", mE_x, mE_y);

    printf("\n");
    printf("Matrix compression dims: x - %i\t y - %i\n", mD_x, mD_y);

    printf("\n");

    //Malloc in cuda for passing matrixes
    hipMalloc((void**)&d_matrizE,sizeof(float)*mE_x*mE_y);
    hipMalloc((void**)&d_result,sizeof(float)*r_x*r_y);

    //Copy pointers of matrix e & d

    hipMemcpy(d_matrizE, matrizE,sizeof(float)*mE_x*mE_y, hipMemcpyHostToDevice);

    //Define sizes fro cuda 
    dim3 Blocks(NumBlocks,NumBlocks);
    dim3 Threads(ThreadsPerBlock,ThreadsPerBlock);
    
    //Take time for the cuda execution
    clock_t begin = clock();

    //Call Cuda global
	valid_compression<<<Blocks, Threads>>>(d_matrizE, mE_x, mE_y, mD_x, mD_y, d_result, r_x, r_y);
    clock_t end = clock();

    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    //Copy result
    hipMemcpy(result, d_result,sizeof(float)*r_x*r_y, hipMemcpyDeviceToHost);

    printf("\n");
    printf("Resultant matrix:\n");
    write_file(result, r_x, r_y);
    printf("\nTime spent: %f \n", time_spent);



    //Free all pointers
    free(matrizE);
    free(result);
    
    //Free all pointers on device
    hipFree(d_matrizE);
    hipFree(d_result);

}

